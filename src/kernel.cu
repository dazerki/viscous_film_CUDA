#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "kernel.h"

__global__ void flux_block(float *u, float* data_3D_gpu, float* data_edge_gpu, float* flx_x, float* flx_y, int nx){
	//position in the grid
	int pos_x = 4*(blockIdx.x * blockDim.x + threadIdx.x)-2;
	int pos_y = 4*(blockIdx.y * blockDim.y + threadIdx.y)-2;

	int pos_data3D_x = 4*(blockIdx.x * blockDim.x + threadIdx.x)-1;
	int pos_data3D_y = 4*(blockIdx.y * blockDim.y + threadIdx.y)-1;



	int pos_x_data = 4*(blockIdx.x * blockDim.x + threadIdx.x);
	int pos_y_data = 4*(blockIdx.y * blockDim.y + threadIdx.y);

	int pos_block_i = threadIdx.x * 4 + 2;
	int pos_block_j = threadIdx.y * 4 + 2;

	int pos_data3D_i = 4* threadIdx.x+1;
	int pos_data3D_j = 4*threadIdx.y+1;

	int pos_block_x = threadIdx.x * 4;
	int pos_block_y = threadIdx.y * 4;

	int size_u_line = (4*blockDim.x+3);
	int size_data_line = 4*blockDim.x+1;
	int size_edge_line = 4*blockDim.x;

	//access memory
	__shared__ float u_local[1225]; //4*dimX+3) * (4*dimY+3
	__shared__ float data_3D_local[3267]; // 3*(4*dimX+1)*(dimY*4+1)
	__shared__ float data_edge_local[2178]; // 2*(4*dimX+1)*(dimY*4+1)
	//u local
	if(threadIdx.x == 0){
		for(int j=1; j<7; j++){
			for(int i=0; i<2; i++){
				u_local[size_u_line*(pos_block_y+j) + (i+pos_block_x)] = u[((pos_y+j+nx)%nx)*nx + ((pos_x+i+nx)%nx)];
			}
		}
	}
	if (threadIdx.x == blockDim.x-1){
		for(int j=2; j<6; j++){
			u_local[size_u_line*(pos_block_y+j) + (6+pos_block_x)] = u[((pos_y+j+nx)%nx)*nx + ((pos_x+6+nx)%nx)];
		}
	}
	if(threadIdx.y == 0){
		for(int j=0; j<2; j++){
			for(int i=2; i<6; i++){
				u_local[size_u_line*(pos_block_y+j) + (i+pos_block_x)] = u[((pos_y+j+nx)%nx)*nx + ((pos_x+i+nx)%nx)];
			}
		}

	}
	if (threadIdx.y == blockDim.y-1){
		for(int i=2; i<6; i++){
			u_local[size_u_line*(pos_block_y+6) + (i+pos_block_x)] = u[((pos_y+6+nx)%nx)*nx + ((pos_x+i+nx)%nx)];
		}
	}


	for(int j=2; j<6; j++){
		for(int i=2; i<6; i++){
			u_local[size_u_line*(pos_block_y+j) + (i+pos_block_x)] = u[(pos_y+j)*nx + (pos_x+i)];
		}
	}

	//data_3D
	if(threadIdx.x == 0){
		for(int j=1; j<5; j++){
			data_3D_local[(size_data_line*(pos_block_y+j) + (pos_block_x))*3] = data_3D_gpu[(((pos_data3D_y+j+nx)%nx)*nx + ((pos_data3D_x+nx)%nx))*3];
			data_3D_local[(size_data_line*(pos_block_y+j) + (pos_block_x))*3 + 1] = data_3D_gpu[(((pos_data3D_y+j+nx)%nx)*nx + ((pos_data3D_x+nx)%nx))*3 + 1];
			data_3D_local[(size_data_line*(pos_block_y+j) + (pos_block_x))*3 + 2] = data_3D_gpu[(((pos_data3D_y+j+nx)%nx)*nx + ((pos_data3D_x+nx)%nx))*3 + 2];
		}

	}
	if(threadIdx.y == 0){
		for(int i=1; i<5; i++){
			data_3D_local[(size_data_line*(pos_block_y) + (pos_block_x + i))*3] = data_3D_gpu[(((pos_data3D_y+nx)%nx)*nx + ((pos_data3D_x + i+nx)%nx))*3];
			data_3D_local[(size_data_line*(pos_block_y) + (pos_block_x + i))*3 + 1] = data_3D_gpu[(((pos_data3D_y+nx)%nx)*nx + ((pos_data3D_x + i+nx)%nx))*3 + 1];
			data_3D_local[(size_data_line*(pos_block_y) + (pos_block_x + i))*3 + 2] = data_3D_gpu[(((pos_data3D_y+nx)%nx)*nx + ((pos_data3D_x + i+nx)%nx))*3 + 2];
		}

	}

	for(int j=1; j<5; j++){
		for(int i=1; i<5; i++){
			data_3D_local[(size_data_line*(pos_block_y+j) + (i+pos_block_x))*3] = data_3D_gpu[((pos_data3D_y+j)*nx + (pos_data3D_x+i))*3];
			data_3D_local[(size_data_line*(pos_block_y+j) + (i+pos_block_x))*3 + 1] = data_3D_gpu[((pos_data3D_y+j)*nx + (pos_data3D_x+i))*3 + 1];
			data_3D_local[(size_data_line*(pos_block_y+j) + (i+pos_block_x))*3 + 2] = data_3D_gpu[((pos_data3D_y+j)*nx + (pos_data3D_x+i))*3 + 2];
		}
	}

	//data_edge
	for(int j=0; j<4; j++){
		for(int i=0; i<4; i++){
			data_edge_local[(size_edge_line*(pos_block_y+j) + (i+pos_block_x))*2] = data_edge_gpu[((pos_y_data+j)*nx + (pos_x_data+i))*2];
			data_edge_local[(size_edge_line*(pos_block_y+j) + (i+pos_block_x))*2 + 1] = data_edge_gpu[((pos_y_data+j)*nx + (pos_x_data+i))*2 + 1];
		}
	}
	__syncthreads();

	float W_q, W_p, M, theta, f, delta_u, lap_p, lap_q;
	float H_p, H_q, T_p, T_q, ct_p, ct_q;
	float k_E, H_E;
	int i_p, j_p;
	int di, dj;

	float u_p, u_q;
	float h = 1.0f/nx;

	float tau = 0.0002f ;
	float e = 0.01f;
	float eta = 0.005f;
	float G = 5.0f;
	float beta = 0.0f;

	for(int direction=0; direction<2; direction++){
		if(direction==0){ //horizontal
			di = 1;
			dj = 0;
		} else { //vertical
			di = 0;
			dj = 1;
		}
		for(int i=0; i<4; i++){
			for(int j=0; j<4; j++){
				i_p = i - di;
				j_p = j - dj;

				if(direction == 0){
					lap_q = (u_local[size_u_line*(pos_block_j+j) + (i+pos_block_i+1)] + u_local[size_u_line*(pos_block_j+j+1) + (i+pos_block_i)] + u_local[size_u_line*(pos_block_j+j-1) + (i+pos_block_i)]);
					lap_p = (u_local[size_u_line*(pos_block_j+j_p) + (i_p+pos_block_i-1)] + u_local[size_u_line*(pos_block_j+j_p+1) + (i_p+pos_block_i)] + u_local[size_u_line*(pos_block_j+j_p-1) + (i_p+pos_block_i)]);
					// if(lap_p-lap_q>0.1 || lap_q - lap_p >0.1){
					// 	printf("HERE (i,j) = (%d,%d), u_i-1 = %f, u_j+1 = %f, u_j-1 = %f\n", i_p+pos_block_i,j_p+pos_block_j, u_local[size_u_line*(pos_block_j+j_p) + (i_p+pos_block_i-1)], u_local[size_u_line*(pos_block_j+j_p+1) + (i_p+pos_block_i)], u_local[size_u_line*(pos_block_j+j_p-1) + (i_p+pos_block_i)]);
					// }
				} else {
					lap_q = (u_local[size_u_line*(pos_block_j+j) + (i+pos_block_i+1)] + u_local[size_u_line*(pos_block_j+j+1) + (i+pos_block_i)] + u_local[size_u_line*(pos_block_j+j) + (i+pos_block_i-1)]);
					lap_p = (u_local[size_u_line*(pos_block_j+j_p) + (i_p+pos_block_i-1)] + u_local[size_u_line*(pos_block_j+j_p) + (i_p+pos_block_i+1)] + u_local[size_u_line*(pos_block_j+j_p-1) + (i_p+pos_block_i)]);
				}



				u_p = u_local[size_u_line*(pos_block_j+j_p) + (i_p+pos_block_i)];
				u_q = u_local[size_u_line*(pos_block_j+j) + (i+pos_block_i)];

				H_p = data_3D_local[(size_data_line*(pos_data3D_j+j_p) + (i_p+pos_data3D_i))*3];
				H_q = data_3D_local[(size_data_line*(pos_data3D_j+j) + (i+pos_data3D_i))*3];

				T_p = data_3D_local[(size_data_line*(pos_data3D_j+j_p) + (i_p+pos_data3D_i))*3 + 1];
				T_q = data_3D_local[(size_data_line*(pos_data3D_j+j) + (i+pos_data3D_i))*3 + 1];

				ct_p = data_3D_local[(size_data_line*(pos_data3D_j+j_p) + (i_p+pos_data3D_i))*3 + 2];
				ct_q = data_3D_local[(size_data_line*(pos_data3D_j+j) + (i+pos_data3D_i))*3 + 2];

				k_E = data_edge_local[(size_edge_line*(pos_block_y+j) + (i+pos_block_x))*2];
				H_E = data_edge_local[(size_edge_line*(pos_block_y+j) + (i+pos_block_x))*2 + 1];

				W_q = G*(nx-(pos_y_data+j)-0.5f)*h - H_q;
				W_p = G*(nx-(pos_y_data+j_p)-0.5f)*h - H_p;




				M = 2.0f * u_q*u_q * u_p*u_p /(3.0f*(u_q + u_p)) + (e/6.0f)*u_q*u_q*u_p*u_p*(H_E+k_E) + (beta/2.0f)*(u_p*u_p + u_q*u_q);

				theta = h*h + (tau*M*(4.0f*e + 2.0f*eta + G*e*(ct_p + ct_q) - e*(T_p + T_q)));
				f = (M*h/(theta)) * (eta*(u_p - u_q) + (e/2.0f)*(lap_q - lap_p + 5.0f*(u_p-u_q)) + W_p-W_q + e*((G*ct_q - T_q)*u_q - (G*ct_p - T_p)*u_p));


				float val = tau*f/h;
				if(u_p<val){
					if(u_p > -u_q){
						delta_u = u_p;
					} else {
						delta_u = -u_q;
					}
				} else{
					if(val > -u_q){
						delta_u = val;

					} else {
						delta_u = -u_q;
					}
				}

				if(direction==0){
						flx_x[(pos_y_data+j)*nx + (pos_x_data+i)] = delta_u;
				} else {
						flx_y[(pos_y_data+j)*nx + (pos_x_data+i)] = delta_u;
				}

			}
		}
	}

}

__global__ void update_u(float *u, float* flux, int dir, int flag, int nx){
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int i = k%nx;
	int j = k/nx;
	int i_p, j_p;

	if(dir==0){ //horizontal
		i_p = i-1;
		j_p = j;
	} else { //vertical
		i_p = i;
		j_p = j-1;
	}
	if((i+(nx+1)*j)%2 == flag){
		u[k] += flux[k];
		u[nx*((j_p+nx)%nx)+((i_p+nx)%nx)] -= flux[k];
		flux[k] = 0.0f;
	}



}
