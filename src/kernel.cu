#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


#include "kernel.h"

__global__ void flux_x(float *u, int rho)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
	// int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
  int rho_ij;
	int nx = 512;
	int ny = 512;
	int di = 1;
	int dj = 0;


	int i = (int) k % nx;
	int j = (int) k / nx;

	rho_ij = ((dj+1)*i + (di+1)*j + rho) % 4;

	if (rho_ij == 3){

		int i_p, j_p;
		float W_q, W_p, M, theta, f, delta_u, lap_p, lap_q;
		float u_p, u_q;
	  float h = 1.0f/nx;

	  float tau = 0.001f ;
		float e = 0.01f;
		float eta = 0.00f;
		float G = 13.0f;
		if (i==0){
			i_p = nx - 1;
			j_p = j - dj;
		} else{
			i_p = i - di;
			j_p = j - dj;
		}

		if (i==nx-1){
			if(j==0){
				lap_q = (u[nx*j] + u[nx*(j+1) + i] + u[nx*(ny-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p+1) + i_p] + u[nx*(ny-1) + i_p]);
			} else if(j==ny-1){
				lap_q = (u[nx*j] + u [i] + u[nx*(j-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[i_p] + u[nx*(j_p-1) + i_p]);
			}
			else{
				lap_q = (u[nx*j + (0)] + u[nx*(j+1) + i] + u[nx*(j-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p+1) + i_p] + u[nx*(j_p-1) + i_p]);
			}
		} else if(i==1){
			if(j==0){
				lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(ny-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p+1) + i_p] + u[nx*(ny-1) + i_p]);
			} else if(j==ny-1){
				lap_q = (u[nx*j + (i+1)] + u[nx*(0) + i] + u[nx*(j-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(0) + i_p] + u[nx*(j_p-1) + i_p]);
			}
			else{
				lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(j-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p+1) + i_p] + u[nx*(j_p-1) + i_p]);
			}
		} else if (j==0){
				lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(ny-1) + i]);
				lap_p = (u[nx*j_p + (nx-1)] + u[nx*(j_p+1) + i_p] + u[nx*(ny-1) + i_p]);
		} else if (j==ny-1){
				lap_q = (u[nx*j + (i+1)] + u [i] + u[nx*(j-1) + i]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[i_p] + u[nx*(j_p-1) + i_p]);
		} else{
			lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(j-1) + i]);
			lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p+1) + i_p] + u[nx*(j_p-1) + i_p]);
		}


		u_p = u[nx*j_p + i_p];
		u_q = u[nx*j + i];

		// W_q = G*(ny-j-0.5f)*h;
		// W_p = G*(ny-j_p-0.5f)*h;

		M = 2.0f * u_p*u_p * u_q*u_q /(3.0f*(u_q + u_p));

		theta = h*h + (2.0f*tau*M*(5.0f*e + eta));
		f = (M*h/(theta)) * ((eta+5.0f*e)*(u_p - u_q) + (e)*(lap_q - lap_p));

		float val = tau*f/h;
		if(u_p<val){
			if(u_p > -u_q){
				delta_u = u_p;
			} else {
				delta_u = -u_q;
			}
		} else{
			if(val > -u_q){
				delta_u = val;
			} else {
				delta_u = -u_q;
			}
		}

		if(i == 0){
			u[nx*(j+nx)%nx + (i+nx)%nx] += delta_u;
			u[nx*(j_p+nx)%nx + (i_p+nx)%nx] -= delta_u;
		} else {
			u[nx*j + i] += delta_u;
			u[nx*j_p + i_p] -= delta_u;
		}


	}
}

__global__ void flux_y(float *u, int rho)
{
	//int k = blockIdx.x * blockDim.x + threadIdx.x;
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
	// int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
  int rho_ij;
	int nx = 512;
	int ny = 512;
	int di = 0;
	int dj = 1;


	int i = (int) k % nx;
	int j = (int) k / nx;

	rho_ij = ((dj+1)*i + (di+1)*j + rho) % 4;
	if (rho_ij == 3){

		float W_q, W_p, M, theta, f, delta_u, lap_p, lap_q;
		int i_p, j_p;

		float u_p, u_q;
	  float h = 1.0f/nx;

	  float tau = 0.001f ;
		float e = 0.01f;
		float eta = 0.00f;
		float G = 13.0f;

		if (j==0){
			i_p = i - di;
			j_p = ny - 1;
		} else {
			i_p = i - di;
			j_p = j - dj;
		}


		if (j==ny-1){
			if(i==0){
				lap_q = (u[nx*j + (i+1)] + u[nx*(0) + i] + u[nx*(j) + nx-1]);
				lap_p = (u[nx*j_p + (nx-1)] + u[nx*(j_p) + i_p+1] + u[nx*(j_p-1) + i_p]);
			} else if (i==nx-1){
				lap_q = (u[nx*j + (0)] + u[nx*(0) + i] + u[nx*(j) + i-1]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p) + 0] + u[nx*(j_p-1) + i_p]);
			} else {
				lap_q = (u[nx*j + (i+1)] + u[nx*(0) + i] + u[nx*(j) + i-1]);
				lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p) + i_p+1] + u[nx*(j_p-1) + i_p]);
			}
		} else if (j==1){
			if(i==0){
				lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(j) + nx-1]);
				lap_p = (u[nx*j_p + (nx-1)] + u[nx*(j_p) + i_p+1] + u[nx*(ny-1) + i_p]);
			} else if(i==nx-1){
				lap_q = (u[nx*j + (0)] + u[nx*(j+1) + i] + u[nx*(j) + i-1]);
				lap_p = (u[nx*j_p + (i-1)] + u[nx*(j_p) + 0] + u[nx*(ny-1) + i_p]);
			} else {
				lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(j) + i-1]);
				lap_p = (u[nx*j_p + (i-1)] + u[nx*(j_p) + i_p+1] + u[nx*(ny-1) + i_p]);
			}
		} else if (i==0){
			lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(j) + nx-1]);
			lap_p = (u[nx*j_p + (nx-1)] + u[nx*(j_p) + i_p+1] + u[nx*(j_p-1) + i_p]);
		} else if (i==nx-1){
			lap_q = (u[nx*j + (0)] + u[nx*(j+1) + i] + u[nx*(j) + i-1]);
			lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p) + 0] + u[nx*(j_p-1) + i_p]);
		} else{
			lap_q = (u[nx*j + (i+1)] + u[nx*(j+1) + i] + u[nx*(j) + i-1]);
			lap_p = (u[nx*j_p + (i_p-1)] + u[nx*(j_p) + i_p+1] + u[nx*(j_p-1) + i_p]);
		}

		u_p = u[nx*j_p + i_p];
		u_q = u[nx*j + i];

		W_q = G*(ny-j-0.5f)*h;

		if(j==0){
			W_p = G*(ny-(-1.0f)-0.5f)*h;
		}else{
			W_p = G*(ny-j_p-0.5f)*h;
		}

		M = 2.0f * u_q*u_q * u_p*u_p /(3.0f*(u_q + u_p));

		theta = h*h + (2.0f*tau*M*(5.0f*e + eta));
		f = (M*h/(theta)) * ((eta+5.0f*e)*(u_p - u_q) + (e)*(lap_q - lap_p) + W_p-W_q);

		float val = tau*f/h;
		if(u_p<val){
			if(u_p > -u_q){
				delta_u = u_p;
			} else {
				delta_u = -u_q;
			}
		} else{
			if(val > -u_q){
				delta_u = val;
			} else {
				delta_u = -u_q;
			}
		}

		if(j == 0){
			u[nx*((j+nx)%nx) + ((i+nx)%nx)] += delta_u;
			u[nx*((j_p+nx)%nx) + ((i_p+nx)%nx)] -= delta_u;
		} else {
			u[nx*j + i] += delta_u;
			u[nx*j_p + i_p] -= delta_u;
		}
  }
}
