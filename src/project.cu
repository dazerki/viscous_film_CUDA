
#include "window.h"
#include "shaders.h"
#include "viscous.h"
#include "kernel.h"

#define GLEW_STATIC
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <omp.h>
#include <time.h>
#include <hip/hip_runtime.h>



#define GRID_SIZE 512
#define BLOCK_SIZE 16

int parity(int di, int dj, int i, int j, int rho);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char *argv[]){

	int nx = 512;
	int ny = 512;
	float h = 1.0f/nx ;
	int size = nx*ny;
	int size_x = (nx+1)*(ny);
	int size_y = nx*(ny+1);
  int size_3D = 3*size;

	// memory allocation
	float* u = (float*)calloc(size, sizeof(float));
	float* data_3D = (float*)calloc(size_3D, sizeof(float));
	float* height_center = (float*)calloc(size, sizeof(float));
	float* height_x_edge = (float*)calloc(size_x, sizeof(float));
	float* height_y_edge = (float*)calloc(size_y, sizeof(float));
	float* data_edge_x = (float*)calloc(2*size_x, sizeof(float));
	float* data_edge_y = (float*)calloc(2*size_y, sizeof(float));


	char fileName[] = "../src/brick_fines.txt";

	float *u_gpu, *data_3D_gpu, *data_edge_x_gpu, *data_edge_y_gpu;

  size_t memSize = size*sizeof(float);
  size_t memSize_3D = size_3D*sizeof(float);

	hipMalloc( (void**)&u_gpu, memSize );
	hipMalloc( (void**)&data_3D_gpu, memSize_3D );
	hipMalloc( (void**)&data_edge_x_gpu, 2*size_x*sizeof(float) );
	hipMalloc( (void**)&data_edge_y_gpu, 2*size_y*sizeof(float) );

	//init
	initialization(u, nx, ny, h, 3);
	read_txt(height_center, height_x_edge, height_y_edge, fileName, nx);
	init_surface_height_map(data_3D, height_center, nx, ny, h);
	init_height_map_edge(data_edge_x, data_edge_y, height_x_edge, height_y_edge, nx, ny, h);



	hipMemcpy( u_gpu, u, memSize, hipMemcpyHostToDevice );
	hipMemcpy( data_3D_gpu, data_3D, memSize_3D, hipMemcpyHostToDevice );
	hipMemcpy( data_edge_x_gpu, data_edge_x, 2*size_x*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( data_edge_y_gpu, data_edge_y, 2*size_y*sizeof(float), hipMemcpyHostToDevice );

  int Nblocks = (nx*nx + 255)/256;
  int Nthreads = 256;

  // Initialise window
  GLFWwindow *window = init_window();

  // Initialise shaders
  init_shaders();

  // Create Vertex Array Object
  GLuint vao;
  glGenVertexArrays(1, &vao);
  glBindVertexArray(vao);

  // Create a Vertex Buffer Object for positions
  GLuint vbo_pos;
  glGenBuffers(1, &vbo_pos);

	GLfloat positions[2*nx*nx];
  for (int i = 0; i < nx; i++) {
      for (int j = 0; j < nx; j++) {
          int ind = j*nx+i;
          positions[2*ind  ] = (float)(1.0 - 2.0*i/(nx-1));
          positions[2*ind+1] = (float)(1.0 - 2.0*j/(nx-1));
      }
  }

  glBindBuffer(GL_ARRAY_BUFFER, vbo_pos);
  glBufferData(GL_ARRAY_BUFFER, sizeof(positions), positions, GL_STATIC_DRAW);

  // Specify vbo_pos' layout
  GLint posAttrib = glGetAttribLocation(shaderProgram, "position");
  glEnableVertexAttribArray(posAttrib);
  glVertexAttribPointer(posAttrib, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);

  // Create an Element Buffer Object and copy the element data to it
  GLuint ebo;
  glGenBuffers(1, &ebo);

	GLuint elements[4*(nx-1)*(nx-1)];
    for (int i = 0; i < nx-1; i++) {
        for (int j = 0; j < nx-1; j++) {
            int ind  = i*nx+j;
            int ind_ = i*(nx-1)+j;

            elements[4*ind_  ] = ind;
            elements[4*ind_+1] = ind+1;
            elements[4*ind_+2] = ind+nx;
            elements[4*ind_+3] = ind+nx+1;
        }
    }

  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, ebo);
  glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(elements), elements, GL_STATIC_DRAW);

	// Create a Vertex Buffer Object for colors
  GLuint vbo_colors;
  glGenBuffers(1, &vbo_colors);

  GLfloat colors[nx*nx];
  for (int i = 0; i < nx; i++) {
      for (int j = 0; j < nx; j++) {
          int ind = i*nx+j;
          colors[ind] = (float) u[ind];
      }
  }

  glBindBuffer(GL_ARRAY_BUFFER, vbo_colors);
  glBufferData(GL_ARRAY_BUFFER, sizeof(colors), colors, GL_STREAM_DRAW);

  // Specify vbo_color's layout
  GLint colAttrib = glGetAttribLocation(shaderProgram, "color");
  glEnableVertexAttribArray(colAttrib);
  glVertexAttribPointer(colAttrib, 1, GL_FLOAT, GL_FALSE, 0, (void*)0);

	// PARAMETER
	float tau = 0.001f ;
	int n_passe = 100;

  // struct timeval start, end;
  // gettimeofday(&start, NULL);


	//LOOP IN TIME
  while(!glfwWindowShouldClose(window)) {
  	for(int p=0; p<n_passe; p++){
  		for(int rho=0; rho<4; rho++){
  			flux_x<<<Nblocks, Nthreads>>>(u_gpu, data_3D_gpu, data_edge_x_gpu, 1, 0, rho);
  		}

  		for(int rho=0; rho<4; rho++){
  			flux_y<<<Nblocks, Nthreads>>>(u_gpu, data_3D_gpu, data_edge_y_gpu, 0, 1, rho);
  		}
  	}

  	hipMemcpy( u, u_gpu, size*sizeof(float), hipMemcpyDeviceToHost );

    glfwSwapBuffers(window);
  	glfwPollEvents();

  	// Clear the screen to black
  	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
  	glClear(GL_COLOR_BUFFER_BIT);

  	for (int i = 0; i < nx*nx; i++) {
  			colors[i] = (float) (u[i]);
  	}

  	glBindBuffer(GL_ARRAY_BUFFER, vbo_colors);
  	glBufferData(GL_ARRAY_BUFFER, sizeof(colors), colors, GL_STREAM_DRAW);


  	// Draw elements
  	glDrawElements(GL_LINES_ADJACENCY, 4*(nx-1)*(nx-1), GL_UNSIGNED_INT, 0);

  	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
  			glfwSetWindowShouldClose(window, GL_TRUE);

  }

  // gettimeofday(&end, NULL);
  //
  // double delta = ((end.tv_sec  - start.tv_sec) * 1000000u +
  //        end.tv_usec - start.tv_usec) / 1.e6;
  // printf("Time taken: %f \n", delta);


	//free memory
	free(u);
	free(data_3D);
	free(height_center);
	free(height_x_edge); free(height_y_edge);
	free(data_edge_x); free(data_edge_y);

	hipFree(u_gpu);
  hipFree(data_3D_gpu);
  hipFree(data_edge_x_gpu);
  hipFree(data_edge_y_gpu);

	printf("\n *Happy computer sound* \n");

	return 0;
}


int parity(int di, int dj, int i, int j, int rho){
	return ((dj+1)*i + (di+1)*j + rho) % 4;
}
